#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <vector>

#include <stdio.h>

static inline void check(hipError_t err, const char *context) {
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA error at %s: %s\n", context, hipGetErrorString(err));
    std::exit(EXIT_FAILURE);
  }
}

#define CHECK(x) check(x, #x)

__global__ void saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    y[i] = a * x[i] + y[i];
}

int main(void) {
  setbuf(stderr, NULL);
  fprintf(stderr, "Start\n");

  int rtVersion, driverVersion;
  CHECK(hipRuntimeGetVersion(&rtVersion));
  CHECK(hipDriverGetVersion(&driverVersion));

  fprintf(stderr, "Runtime version: %d\n", rtVersion);
  fprintf(stderr, "Driver version: %d\n", driverVersion);

  constexpr int N = 1 << 10;

  std::vector<float> xHost(N), yHost(N);
  for (int i = 0; i < N; i++) {
    xHost[i] = 1.0f;
    yHost[i] = 2.0f;
  }

  fprintf(stderr, "Host memory initialized, copying to the device\n");
  fflush(stderr);

  float *xDevice, *yDevice;
  CHECK(hipMalloc(&xDevice, N * sizeof(float)));
  CHECK(hipMalloc(&yDevice, N * sizeof(float)));

  CHECK(hipMemcpy(xDevice, xHost.data(), N * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(yDevice, yHost.data(), N * sizeof(float),
                   hipMemcpyHostToDevice));
  fprintf(stderr, "Scheduled a hipMemcpy, calling the kernel\n");

  saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, xDevice, yDevice);
  fprintf(stderr, "Scheduled a kernel call\n");
  CHECK(hipGetLastError());

  CHECK(hipMemcpy(yHost.data(), yDevice, N * sizeof(float),
                   hipMemcpyDeviceToHost));

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(yHost[i] - 4.0f));
  fprintf(stderr, "Max error: %f\n", maxError);

  CHECK(hipFree(xDevice));
  CHECK(hipFree(yDevice));
}
